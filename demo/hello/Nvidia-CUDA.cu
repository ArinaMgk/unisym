﻿// ASCII CPP-NVCC
// : test on Visual Studio 2022
// $ nvcc kernel.cu -o ciallo # you may need this for Linux

// Optional if the suffix of the document is `.cu`
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void ciallo()
{
	printf("Ciallo~ CUDA~ @dosconio, 20240418\n");
}

int main() 
{
	ciallo <<<2, 2>>> ();// the `ciallo` will be called for 2*2 times!
	hipDeviceSynchronize();
}

